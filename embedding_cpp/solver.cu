#include "hipsparse.h"
#include "solver.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "timer.h"

namespace solver {
std::unique_ptr<double> power_iteration(const CSR<double> &cooccurrence,
                                        const size_t n_iterations,
                                        const size_t n_dimensions) {
  auto cuda_init = timer::start_clock();
  hipsparseHandle_t handle = 0;
  hipsparseStatus_t status;
  hipsparseMatDescr_t descr = 0;
  /* initialize cusparselibrary */
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    // TODO Throw error: CLEANUP("CUSPARSE Library initialization failed");
    return std::unique_ptr<double>();
  }
  /* create and setup matrix descriptor */
  status = hipsparseCreateMatDescr(&descr);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    // TODO Throw error: CLEANUP("Matrix descriptor initialization failed");
    return std::unique_ptr<double>();
  }
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  double *cuda_val;
  int *cuda_row_ptr;
  int *cuda_col_ind;
  hipMalloc(&cuda_val, cooccurrence.nnz * sizeof(double));
  hipMalloc(&cuda_row_ptr, (cooccurrence.n + 1) * sizeof(int));
  hipMalloc(&cuda_col_ind, cooccurrence.nnz * sizeof(int));

  hipMemcpy(cuda_val, cooccurrence.val, cooccurrence.nnz * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(cuda_row_ptr, cooccurrence.rowPtr, (cooccurrence.n + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(cuda_col_ind, cooccurrence.colInd, cooccurrence.nnz * sizeof(int),
             hipMemcpyHostToDevice);

  double *embedding;
  hipMalloc(&embedding, cooccurrence.n * n_dimensions * sizeof(double));
  double *x;
  hipMalloc(&x, cooccurrence.n * n_dimensions * sizeof(double));

  timer::stop_clock("CUDA INIT",cuda_init);


  for (size_t i = 0; i < n_iterations; i++) {
    auto itr_timer = timer::start_clock();
    // C = α ∗ op(A) ∗ B + β ∗ C
    const double alpha = 1.0;
    const double beta = -1.0;
    hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cooccurrence.n,
                   n_dimensions, cooccurrence.n, cooccurrence.nnz,
                   &alpha, descr, cuda_val, cuda_row_ptr,
                   cuda_col_ind, embedding, n_dimensions,
                   &beta, x, n_dimensions);
    timer::stop_clock("ITERATION",itr_timer);
  }

  hipFree(cuda_val);
  hipFree(cuda_row_ptr);
  hipFree(cuda_col_ind);

  double *_x = (double *)malloc(sizeof(double) * cooccurrence.n * n_dimensions);
  hipMemcpy(_x, x, cooccurrence.n * n_dimensions * sizeof(double),
             hipMemcpyDeviceToHost);
  return std::unique_ptr<double>(_x);
}
} // en