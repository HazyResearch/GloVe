#include "hipsparse.h"
#include "solver.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "timer.h"

namespace solver {
std::unique_ptr<double> power_iteration(const CSR<double> &cooccurrence,
                                        const size_t n_iterations,
                                        const size_t n_dimensions) {
  auto cuda_init = timer::start_clock();
  hipsparseHandle_t handle = 0;
  hipsparseStatus_t status;
  hipsparseMatDescr_t descr = 0;
  /* initialize cusparselibrary */
  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    // TODO Throw error: CLEANUP("CUSPARSE Library initialization failed");
    return std::unique_ptr<double>();
  }
  /* create and setup matrix descriptor */
  status = hipsparseCreateMatDescr(&descr);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    // TODO Throw error: CLEANUP("Matrix descriptor initialization failed");
    return std::unique_ptr<double>();
  }
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  double *cuda_val;
  int *cuda_row_ptr;
  int *cuda_col_ind;
  hipMalloc(&cuda_val, cooccurrence.nnz * sizeof(double));
  hipMalloc(&cuda_row_ptr, (cooccurrence.n + 1) * sizeof(int));
  hipMalloc(&cuda_col_ind, cooccurrence.nnz * sizeof(int));

  hipMemcpy(cuda_val, cooccurrence.val, cooccurrence.nnz * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(cuda_row_ptr, cooccurrence.rowPtr, (cooccurrence.n + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(cuda_col_ind, cooccurrence.colInd, cooccurrence.nnz * sizeof(int),
             hipMemcpyHostToDevice);

  double *embedding;
  hipMalloc(&embedding, cooccurrence.n * n_dimensions * sizeof(double));
  double *x;
  hipMalloc(&x, cooccurrence.n * n_dimensions * sizeof(double));

  timer::stop_clock("CUDA INIT",cuda_init);

  std::cout << cooccurrence.n << " " << cooccurrence.nnz << std::endl;
  std::cout << n_iterations << " " << n_dimensions << std::endl;

  for (size_t i = 0; i < n_iterations; i++) {
    const auto itr_timer = timer::start_clock();
    // C = α ∗ op(A) ∗ B + β ∗ C
    const double alpha = 1.0;
    const double beta = -1.0;
    status = hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cooccurrence.n,
                   n_dimensions, cooccurrence.n, cooccurrence.nnz,
                   &alpha, descr, cuda_val, cuda_row_ptr,
                   cuda_col_ind, embedding, cooccurrence.n,
                   &beta, x, cooccurrence.n);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      std::cout << "ERROR: Matrix-matrix multiplication failed" << std::endl;
      std::cout << "status = " << status << std::endl;
      return std::unique_ptr<double>();
    }  
    hipDeviceSynchronize();
    timer::stop_clock("ITERATION " + std::to_string(i), itr_timer);
  }

  const auto cuda_free = timer::start_clock();
  hipFree(cuda_val);
  hipFree(cuda_row_ptr);
  hipFree(cuda_col_ind);
  timer::stop_clock("CUDA FREE",cuda_free);

  const auto cpu_xfr = timer::start_clock();
  double *_x = (double *)malloc(sizeof(double) * cooccurrence.n * n_dimensions);
  hipMemcpy(_x, x, cooccurrence.n * n_dimensions * sizeof(double),
             hipMemcpyDeviceToHost);
  timer::stop_clock("CPU XFR", cpu_xfr);
  return std::unique_ptr<double>(_x);
}
} // en