#include "utils.h"
#include "timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>

namespace utils {

void sen_test(){

  const size_t alloc_size = 300000000;
  double* big_alloc = (double*)malloc(alloc_size*sizeof(double));


  const auto sen_time = timer::start_clock();
  double* cuda_big_alloc;
  hipMalloc(&cuda_big_alloc, alloc_size);

  hipMemcpy(cuda_big_alloc, big_alloc, alloc_size,
             hipMemcpyHostToDevice);
  timer::stop_clock("SEN TIME", sen_time);
}


}
